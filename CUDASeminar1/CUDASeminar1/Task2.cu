#include "hip/hip_runtime.h"
/*
	������������� ����������� �� ������������� ��������
	COMPARE - ����� ��������� ���������� �� GPU � �� CPU
	PRINT - ����� ������ �������� ��������� ��������
	EXPLICIT - ����� ����� ��������� ����������
	IMPLICIT - ������� ����� ��������� ����������
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>

#define CUDA_CALL(x) do { \
		hipError_t cudaStatus = (x); \
		if (cudaStatus != hipSuccess) { \
			printf("Cuda Failed with error %s\n", hipGetErrorString(cudaStatus)); \
			system("pause"); \
			exit(cudaStatus); \
		} \
	} while (0);

#define PRINT
//#define COMPARE
#define EXPLICIT
//#define IMPLICIT

#ifdef IMPLICIT
#define EPS 1.e-3
#endif

#define BLOCK_SIZE 32

#define a 5
#define F(x, y, t) 10

#define X 500
#define Y 500

#define xPoints (50 + 1)
#define yPoints (50 + 1)
#define tPoints (1000 + 1)

#define DT 1
#define DX X * 1.0 / (xPoints - 1)
#define DY Y * 1.0 / (yPoints - 1)

#ifdef EXPLICIT
__global__ void computeZ(double *z, const int k)
#endif
#ifdef IMPLICIT
__global__ void computeZ(double *z, const int k, double *delta)
#endif
{
	int threadId = threadIdx.x + blockDim.x * blockIdx.x;
	// threadId - ��� ��������������� ��� � ��� �������� ����� ������� ����������� xPoints * yPoints
	if (threadId >= xPoints * yPoints) {
		return;
	}

#ifdef EXPLICIT
	int x = threadId / yPoints;
	int y = threadId % yPoints;
	// ����� �������, ���������� threadId = x * yPoints + y
	// ������� ������� - �� ������� �������� z ����� 0
	if (x == 0 || y == 0 || x == xPoints - 1 || y == yPoints - 1) {
		z[k * xPoints * yPoints + threadId] = 0;
	}
	else {
		z[k * xPoints * yPoints + x * yPoints + y] = DT * DT * (
				F(x, y, k) + a * a * (
					(
						z[(k - 1) * xPoints * yPoints + (x + 1) * yPoints + y]
						- 2 * z[(k - 1) * xPoints * yPoints + x * yPoints + y]
						+ z[(k - 1) * xPoints * yPoints + (x - 1) * yPoints + y]
					) / (DX * DX)
					+
					(
						z[(k - 1) * xPoints * yPoints + x * yPoints + y + 1]
						- 2 * z[(k - 1) * xPoints * yPoints + x * yPoints + y]
						+ z[(k - 1) * xPoints * yPoints + x * yPoints + y - 1]
					) / (DY * DY)
				)
			)
			- z[(k - 2) * xPoints * yPoints + x * yPoints + y]
			+ 2 * z[(k - 1) * xPoints * yPoints + x * yPoints + y];
	}
#endif
#ifdef IMPLICIT
	// ���� ��������
	if (threadId == xPoints - 1) {
		z[k*xPoints + threadId] = z[(k - 1) * xPoints + threadId] + DT;
	}
	else {
		if (threadId < xPoints - 1 && threadId > 0) {
			delta[threadId] = z[k * xPoints + threadId];
			z[k * xPoints + threadId] = (
				z[(k - 1) * xPoints + threadId] +
				DT / (DX * DX) * temp[(k - 1) * xPoints + threadId - 1] +
				DT / (DX * DX) * temp[(k - 1) * xPoints + threadId + 1]
				) / (2 * DT / (DX * DX) + 1);
			delta[threadId] = abs(z[k * xPoints + threadId] - delta[threadId]);
		}
	}
#endif
}

int main()
{
#ifdef COMPARE
	hipEvent_t GPUStartWithMem, GPUStartKernelOnly, GPUStopWithMem, GPUStopKernelOnly;
	float CPUStart, CPUStop;

	float GPUTimeWithMem = 0.0f;
	float GPUTimeKernelOnly = 0.0f;
	float CPUTime = 0.0f;
#endif

	int totalElemCount = xPoints * yPoints * (tPoints + 1); // ����� k - 2. ����� ������������ ������� ��� -1
	int memSize = totalElemCount * sizeof(double);

	double *z = (double *)calloc(totalElemCount, sizeof(double));
	double *devZ;

	CUDA_CALL(hipSetDevice(0));
	CUDA_CALL(hipDeviceReset());
	CUDA_CALL(hipMalloc(&devZ, memSize));

#ifdef IMPLICIT
	double *delta = (double *)calloc(xPoints, sizeof(double));
	double *devDelta;
	CUDA_CALL(hipMalloc(&devDelta, xPoints * sizeof(double)));
#endif

#ifdef COMPARE
	CUDA_CALL(hipEventCreate(&GPUStartKernelOnly));
	CUDA_CALL(hipEventCreate(&GPUStopKernelOnly));
	CUDA_CALL(hipEventCreate(&GPUStartWithMem));
	CUDA_CALL(hipEventCreate(&GPUStopWithMem));

	CUDA_CALL(hipEventRecord(GPUStartWithMem, 0));
#endif

	CUDA_CALL(hipMemcpy(devZ, z, memSize, hipMemcpyHostToDevice));

	int blocksCount = xPoints * yPoints / BLOCK_SIZE;
	if (xPoints * yPoints % BLOCK_SIZE != 0) {
		++blocksCount;
	}

#ifdef COMPARE
	CUDA_CALL(hipEventRecord(GPUStartKernelOnly, 0));
#endif

	for (int k = 2; k < tPoints + 1; ++k) {
#ifdef IMPLICIT
		bool flag = false; // ���� ���������� ������� ����
		while (!flag) {
			computeTemp << <blocksCount, BLOCK_SIZE >> > (devTemp, k, devDelta);
			CUDA_CALL(hipMemcpy(delta, devDelta, xPoints * sizeof(double), hipMemcpyDeviceToHost));
			double sum = 0;
			for (int i = 0; i < xPoints; ++i) {
				sum += delta[i];
			}
			if (sum < EPS) {
				flag = true;
			}
		}
#endif
#ifdef EXPLICIT
		computeZ << <blocksCount, BLOCK_SIZE >> >(devZ, k);
#endif
	}

	CUDA_CALL(hipGetLastError());
	CUDA_CALL(hipDeviceSynchronize());

#ifdef COMPARE
	CUDA_CALL(hipEventRecord(GPUStopKernelOnly, 0));
#endif

	CUDA_CALL(hipMemcpy(z, devZ, memSize, hipMemcpyDeviceToHost));

#ifdef PRINT
	for (int k = 1; k < tPoints + 1; ++k) {
		for (int i = 0; i < xPoints; ++i) {
			for (int j = 0; j < yPoints; ++j) {
				printf("%f %f %f\n", i * DX, j * DY, z[k * xPoints * yPoints + i * yPoints + j]);
			}
		}
		if (k != tPoints) {
			printf("\n\n");
		}
	}
#endif // PRINT

#ifdef COMPARE
	CUDA_CALL(hipEventRecord(GPUStopWithMem, 0));
	CUDA_CALL(hipEventSynchronize(GPUStopWithMem));
	CUDA_CALL(hipEventElapsedTime(&GPUTimeWithMem, GPUStartWithMem, GPUStopWithMem));
	CUDA_CALL(hipEventElapsedTime(&GPUTimeKernelOnly, GPUStartKernelOnly, GPUStopKernelOnly));
	printf("GPU Time With Mem: %.3f ms\n", GPUTimeWithMem);
	printf("GPU Time Kernel Only: %.3f ms\n", GPUTimeKernelOnly);

	CUDA_CALL(hipEventDestroy(GPUStartKernelOnly));
	CUDA_CALL(hipEventDestroy(GPUStopKernelOnly));
	CUDA_CALL(hipEventDestroy(GPUStartWithMem));
	CUDA_CALL(hipEventDestroy(GPUStopWithMem));

	CPUStart = clock();
	memset(temp, 0, memSize);
#ifdef EXPLICIT
	for (int k = 1; k < tPoints; ++k) {
		temp[k*xPoints + xPoints - 1] = temp[(k - 1) * xPoints + xPoints - 1] + DT;
		for (int j = 0; j < xPoints - 1; ++j) {
			temp[k*xPoints + j] = (
				temp[(k - 1) * xPoints + j + 1] -
				2 * temp[(k - 1) * xPoints + j] +
				temp[(k - 1) * xPoints + j - 1]
				) * DT / (DX * DX) + temp[(k - 1) * xPoints + j];
		}
	}
#endif
#ifdef IMPLICIT
	for (int k = 1; k < tPoints; ++k) {
		bool flag = false; // ���� ���������� ������� ����
		while (!flag) {
			temp[k*xPoints + xPoints - 1] = temp[(k - 1) * xPoints + xPoints - 1] + DT;
			for (int j = 1; j < xPoints - 1; ++j) {
				delta[j] = temp[k * xPoints + j];
				temp[k * xPoints + j] = (
					temp[(k - 1) * xPoints + j] +
					DT / (DX * DX) * temp[(k - 1) * xPoints + j - 1] +
					DT / (DX * DX) * temp[(k - 1) * xPoints + j + 1]
					) / (2 * DT / (DX * DX) + 1);
				delta[j] = abs(temp[k * xPoints + j] - delta[j]);
			}
			double sum = 0;
			for (int i = 0; i < xPoints; ++i) {
				sum += delta[i];
			}
			if (sum < EPS) {
				flag = true;
			}
		}
	}
#endif

	CPUStop = clock();
	CPUTime = 1000. * (CPUStop - CPUStart) / CLOCKS_PER_SEC;
	printf("CPU time: %.3f ms\n", CPUTime);

#endif // COMPARE

	CUDA_CALL(hipFree(devZ));
	CUDA_CALL(hipDeviceReset());
	free(z);

	return 0;
}
