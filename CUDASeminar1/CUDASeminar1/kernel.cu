/*
	��1: 
		- gnuplot;
		- ������ ��� CPU + ���������;
		- ������� �2
		*- ������������ ����� ��� ������� �����, � ���� ��������������
*/


#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#include <iostream>

#define CUDA_CALL(x) do { \
		hipError_t cudaStatus = (x); \
		if (cudaStatus != hipSuccess) { \
			printf("Cuda Failed with error %s\n", hipGetErrorString(cudaStatus)); \
			system("pause"); \
			exit(cudaStatus); \
		} \
	} while (0);

#define PRINT
//#define COMPARE

#define BLOCK_SIZE 32

#define T 5
#define L 10

#define xPoints (200 + 1)
#define tPoints (10000 + 1)

#define DT T * 1.0 / (tPoints - 1)
#define DX L * 1.0 / (xPoints - 1)

__global__ void computeTemp(double *temp, const int k)
{
	int threadId = threadIdx.x + blockDim.x * blockIdx.x;
	if (threadId == xPoints - 1) {
		temp[k*xPoints + threadId] = temp[(k - 1) * xPoints + threadId] + DT;
	}
	else if (threadId < xPoints - 1 && threadId > 0) {
		temp[k*xPoints + threadId] = (
				temp[(k - 1) * xPoints + threadId + 1] -
				2 * temp[(k - 1) * xPoints + threadId] +
				temp[(k - 1) * xPoints + threadId - 1]
			) * DT / (DX * DX) + temp[(k - 1) * xPoints + threadId];
	}
}

int main()
{
	int totalElemCount = xPoints * tPoints;
	int memSize = totalElemCount * sizeof(double);

	double *temp = (double *)calloc(totalElemCount, sizeof(double));
	double *devTemp;
	
	CUDA_CALL(hipSetDevice(0));
	CUDA_CALL(hipDeviceReset());
	CUDA_CALL(hipMalloc(&devTemp, memSize));
	CUDA_CALL(hipMemcpy(devTemp, temp, memSize, hipMemcpyHostToDevice));

	int blocksCount = xPoints / BLOCK_SIZE;
	if (xPoints % BLOCK_SIZE != 0) {
		++blocksCount;
	}

	for (int k = 1; k < tPoints; ++k) {
		computeTemp<<<blocksCount, BLOCK_SIZE>>>(devTemp, k);
	}

	CUDA_CALL(hipGetLastError());
	CUDA_CALL(hipDeviceSynchronize());
	CUDA_CALL(hipMemcpy(temp, devTemp, memSize, hipMemcpyDeviceToHost));
	
#ifdef PRINT
	for (int i = 0; i < tPoints; ++i) {
		for (int j = 0; j < xPoints; ++j) {
			printf("%f %f\n", j * DX, temp[i * xPoints + j]);
		}
		if (i != tPoints - 1) {
			printf("\n\n");
		}
}
#endif // PRINT
	
#ifdef COMPARE

#endif // COMPARE


	CUDA_CALL(hipFree(devTemp));
	CUDA_CALL(hipDeviceReset());
	free(temp);

    return 0;
}
