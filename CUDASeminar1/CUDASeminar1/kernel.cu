#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>

#define CUDA_CALL(x) do { \
		hipError_t cudaStatus = (x); \
		if (cudaStatus != hipSuccess) { \
			printf("Cuda Failed with error %s\n", hipGetErrorString(cudaStatus)); \
			system("pause"); \
			exit(cudaStatus); \
		} \
	} while (0);

//#define PRINT
#define COMPARE
#define EXPLICIT
//#define IMPLICIT

#ifdef IMPLICIT
#define EPS 1.e-3
#endif

#define BLOCK_SIZE 32

#define L 1000

#define xPoints (200 + 1)
#define tPoints (10000 + 1)

#define DT 5
#define DX L * 1.0 / (xPoints - 1)

#ifdef EXPLICIT
__global__ void computeTemp(double *temp, const int k)
#endif
#ifdef IMPLICIT
__global__ void computeTemp(double *temp, const int k, double *delta)
#endif
{
	int threadId = threadIdx.x + blockDim.x * blockIdx.x;
#ifdef EXPLICIT
	if (threadId == xPoints - 1) {
		temp[k*xPoints + threadId] = temp[(k - 1) * xPoints + threadId] + DT;
	}
	else if (threadId < xPoints - 1 && threadId > 0) {
		temp[k*xPoints + threadId] = (
				temp[(k - 1) * xPoints + threadId + 1] -
				2 * temp[(k - 1) * xPoints + threadId] +
				temp[(k - 1) * xPoints + threadId - 1]
			) * DT / (DX * DX) + temp[(k - 1) * xPoints + threadId];
	}
#endif
#ifdef IMPLICIT
	// одна итерация
	if (threadId == xPoints - 1) {
		temp[k*xPoints + threadId] = temp[(k - 1) * xPoints + threadId] + DT;
	}
	else {
		if (threadId < xPoints - 1 && threadId > 0) {
			delta[threadId] = temp[k * xPoints + threadId];
			temp[k * xPoints + threadId] = (
				temp[(k - 1) * xPoints + threadId] +
				DT / (DX * DX) * temp[(k - 1) * xPoints + threadId - 1] +
				DT / (DX * DX) * temp[(k - 1) * xPoints + threadId + 1]
				) / (2 * DT / (DX * DX) + 1);
			delta[threadId] = abs(temp[k * xPoints + threadId] - delta[threadId]);
		}
	}
#endif
}

int main()
{
#ifdef COMPARE
	hipEvent_t GPUStartWithMem, GPUStartKernelOnly, GPUStopWithMem, GPUStopKernelOnly;
	float CPUStart, CPUStop;
	
	float GPUTimeWithMem = 0.0f;
	float GPUTimeKernelOnly = 0.0f;
	float CPUTime = 0.0f;
#endif

	int totalElemCount = xPoints * tPoints;
	int memSize = totalElemCount * sizeof(double);

	double *temp = (double *)calloc(totalElemCount, sizeof(double));
	double *devTemp;

	CUDA_CALL(hipSetDevice(0));
	CUDA_CALL(hipDeviceReset());
	CUDA_CALL(hipMalloc(&devTemp, memSize));

#ifdef IMPLICIT
	double *delta = (double *)calloc(xPoints, sizeof(double));
	double *devDelta;
	CUDA_CALL(hipMalloc(&devDelta, xPoints * sizeof(double)));
#endif

#ifdef COMPARE
	CUDA_CALL(hipEventCreate(&GPUStartKernelOnly));
	CUDA_CALL(hipEventCreate(&GPUStopKernelOnly));
	CUDA_CALL(hipEventCreate(&GPUStartWithMem));
	CUDA_CALL(hipEventCreate(&GPUStopWithMem));

	CUDA_CALL(hipEventRecord(GPUStartWithMem, 0));
#endif

	CUDA_CALL(hipMemcpy(devTemp, temp, memSize, hipMemcpyHostToDevice));

	int blocksCount = xPoints / BLOCK_SIZE;
	if (xPoints % BLOCK_SIZE != 0) {
		++blocksCount;
	}

#ifdef COMPARE
	CUDA_CALL(hipEventRecord(GPUStartKernelOnly, 0));
#endif

	for (int k = 1; k < tPoints; ++k) {
#ifdef IMPLICIT
		bool flag = false; // флаг сходимости решения СЛАУ
		while (!flag) {
			computeTemp << <blocksCount, BLOCK_SIZE >> > (devTemp, k, devDelta);
			CUDA_CALL(hipMemcpy(delta, devDelta, xPoints * sizeof(double), hipMemcpyDeviceToHost));
			double sum = 0;
			for (int i = 0; i < xPoints; ++i) {
				sum += delta[i];
			}
			if (sum < EPS) {
				flag = true;
			}
		}
#endif
#ifdef EXPLICIT
		computeTemp << <blocksCount, BLOCK_SIZE >> >(devTemp, k);
#endif
	}

	CUDA_CALL(hipGetLastError());
	CUDA_CALL(hipDeviceSynchronize());

#ifdef COMPARE
	CUDA_CALL(hipEventRecord(GPUStopKernelOnly, 0));
#endif

	CUDA_CALL(hipMemcpy(temp, devTemp, memSize, hipMemcpyDeviceToHost));
	
#ifdef PRINT
	for (int i = 0; i < tPoints; ++i) {
		for (int j = 0; j < xPoints; ++j) {
			printf("%f %f\n", j * DX, temp[i * xPoints + j]);
		}
		if (i != tPoints - 1) {
			printf("\n\n");
		}
}
#endif // PRINT
	
#ifdef COMPARE
	CUDA_CALL(hipEventRecord(GPUStopWithMem, 0));
	CUDA_CALL(hipEventSynchronize(GPUStopWithMem));
	CUDA_CALL(hipEventElapsedTime(&GPUTimeWithMem, GPUStartWithMem, GPUStopWithMem));
	CUDA_CALL(hipEventElapsedTime(&GPUTimeKernelOnly, GPUStartKernelOnly, GPUStopKernelOnly));
	printf("GPU Time With Mem: %.3f ms\n", GPUTimeWithMem);
	printf("GPU Time Kernel Only: %.3f ms\n", GPUTimeKernelOnly);

	CUDA_CALL(hipEventDestroy(GPUStartKernelOnly));
	CUDA_CALL(hipEventDestroy(GPUStopKernelOnly));
	CUDA_CALL(hipEventDestroy(GPUStartWithMem));
	CUDA_CALL(hipEventDestroy(GPUStopWithMem));

	CPUStart = clock();
	memset(temp, 0, memSize);
#ifdef EXPLICIT
	for (int k = 1; k < tPoints; ++k) {
		temp[k*xPoints + xPoints - 1] = temp[(k - 1) * xPoints + xPoints - 1] + DT;
		for (int j = 0; j < xPoints - 1; ++j) {
				temp[k*xPoints + j] = (
					temp[(k - 1) * xPoints + j + 1] -
					2 * temp[(k - 1) * xPoints + j] +
					temp[(k - 1) * xPoints + j - 1]
					) * DT / (DX * DX) + temp[(k - 1) * xPoints + j];
		}
	}
#endif
#ifdef IMPLICIT
	for (int k = 1; k < tPoints; ++k) {
		bool flag = false; // флаг сходимости решения СЛАУ
		while (!flag) {
			temp[k*xPoints + xPoints - 1] = temp[(k - 1) * xPoints + xPoints - 1] + DT;
			for (int j = 1; j < xPoints - 1; ++j) {
				delta[j] = temp[k * xPoints + j];
				temp[k * xPoints + j] = (
					temp[(k - 1) * xPoints + j] +
					DT / (DX * DX) * temp[(k - 1) * xPoints + j- 1] +
					DT / (DX * DX) * temp[(k - 1) * xPoints + j + 1]
					) / (2 * DT / (DX * DX) + 1);
				delta[j] = abs(temp[k * xPoints + j] - delta[j]);
			}
			double sum = 0;
			for (int i = 0; i < xPoints; ++i) {
				sum += delta[i];
			}
			if (sum < EPS) {
				flag = true;
			}
		}		
	}
#endif

	CPUStop = clock();
	CPUTime = 1000. * (CPUStop - CPUStart) / CLOCKS_PER_SEC;
	printf("CPU time: %.3f ms\n", CPUTime);

#endif // COMPARE

	CUDA_CALL(hipFree(devTemp));
	CUDA_CALL(hipDeviceReset());
	free(temp);

    return 0;
}
